#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <atomic>
#include <thread>

// Variável global para controlar quando parar o estresse
std::atomic<bool> stopStress(false);

// Função para obter informações da GPU
void getGPUInfo() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "Nenhuma GPU CUDA disponível." << std::endl;
        return;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);
        std::cout << "Informações da GPU " << device << ":\n";
        std::cout << "  Nome: " << prop.name << std::endl;
        std::cout << "  Memória Global: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Arquitetura: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Núcleos de Processamento: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Frequência: " << prop.clockRate / 1000 << " MHz" << std::endl;
        std::cout << std::endl;
    }
}

// Função para estressar a GPU com operações de cópia de memória
void stressMemory() {
    const int N = 1 << 28;  // Aumentando o número de elementos para uma cópia de memória maior
    float* d_a, * d_b;
    float* h_a = new float[N];
    float* h_b = new float[N];

    // Alocação de memória na GPU
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));

    // Inicializa os dados na memória da CPU
    for (int i = 0; i < N; ++i) {
        h_a[i] = 1.0f;
    }

    while (!stopStress) {
        // Copia os dados para a memória da GPU
        hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, d_a, N * sizeof(float), hipMemcpyDeviceToDevice);  // Cópia entre a GPU
        hipMemcpy(h_b, d_b, N * sizeof(float), hipMemcpyDeviceToHost);  // Cópia de volta para a CPU
    }

    // Libera a memória alocada na GPU
    hipFree(d_a);
    hipFree(d_b);

    delete[] h_a;
    delete[] h_b;
}

// Função para estressar a GPU com operações de memória (alocação e desalocação repetidas)
void stressCopy() {
    const int N = 1 << 28;  // Aumentando o número de elementos para testar memória de forma mais intensiva
    float* d_data;

    while (!stopStress) {
        // Aloca e desaloca memória várias vezes
        hipMalloc((void**)&d_data, N * sizeof(float));
        hipFree(d_data);
    }
}

// Função para estressar a GPU com operações de 3D (realiza cálculos simples em paralelo)
__global__ void kernel3D(float* d_out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 1 << 28) {  // Aumentando o número de threads processados
        d_out[idx] = sinf((float)idx) * cosf((float)idx);  // Simples cálculo trigonométrico
    }
}

void stress3D() {
    const int N = 1 << 28;  // Número de elementos
    float* d_out;

    hipMalloc((void**)&d_out, N * sizeof(float));

    while (!stopStress) {
        // Aumentando a quantidade de blocos e threads
        kernel3D << <(N + 255) / 256, 256 >> > (d_out);
        hipDeviceSynchronize();
    }

    hipFree(d_out);
}

// Função para estressar a GPU com operações de decodificação de vídeo
void stressVideoDecode() {
    const int N = 1 << 28;  // Tamanho dos dados para simular a decodificação
    uint8_t* d_data;

    // Aloca memória na GPU
    hipMalloc((void**)&d_data, N * sizeof(uint8_t));

    while (!stopStress) {
        // Simula o processamento de vídeo decodificado (operando com os dados)
        for (int i = 0; i < N; ++i) {
            d_data[i] = (uint8_t)(i % 255);
        }
    }

    hipFree(d_data);
}

// Função para estressar a GPU com operações de codificação de vídeo
void stressVideoEncode() {
    const int N = 1 << 28;  // Tamanho dos dados para simular a codificação
    uint8_t* d_data;

    // Aloca memória na GPU
    hipMalloc((void**)&d_data, N * sizeof(uint8_t));

    while (!stopStress) {
        // Simula o processamento de vídeo codificado (operando com os dados)
        for (int i = 0; i < N; ++i) {
            d_data[i] = (uint8_t)(i % 255);
        }
    }

    hipFree(d_data);
}

// Função para estressar a GPU com todas as operações
void stressGPU(int level, int device) {
    hipSetDevice(device); // Seleciona a GPU correta

    // Exemplo de adaptação do estresse com base na placa
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "Usando a GPU: " << prop.name << std::endl;

    switch (level) {
    case 1:
        std::cout << "Estressando a GPU com operações 3D..." << std::endl;
        stress3D();
        break;
    case 2:
        std::cout << "Estressando a GPU com cópia de memória..." << std::endl;
        stressCopy();
        break;
    case 3:
        std::cout << "Estressando a GPU com decodificação de vídeo..." << std::endl;
        stressVideoDecode();
        break;
    case 4:
        std::cout << "Estressando a GPU com codificação de vídeo..." << std::endl;
        stressVideoEncode();
        break;
    case 5:
        std::cout << "Estressando a GPU com operações de memória..." << std::endl;
        stressMemory();
        break;
    case 6:
        std::cout << "Estressando a GPU com todas as operações..." << std::endl;
        stress3D();
        stressCopy();
        stressVideoDecode();
        stressVideoEncode();
        stressMemory();
        break;
    default:
        std::cout << "Número inválido. Escolha um número entre 1 e 6." << std::endl;
    }
}

// Função para permitir ao usuário parar o estresse com uma tecla
void stopStressInput() {
    char stop;
    std::cout << "Digite 'q' para parar o estresse: ";
    std::cin >> stop;
    if (stop == 'q') {
        stopStress = true;
    }
}

int main(int argc, char* argv[]) {
    // Exibir as informações da GPU
    getGPUInfo();

    int level = std::atoi(argv[1]);

    if (level < 1 || level > 6) {
        /*std::cout << "Selecione o tipo de teste a ser feito:\n";
        std::cout << "1 - Estressando a GPU com operações 3D\n";
        std::cout << "2 - Estressando a GPU com cópia de memória\n";
        std::cout << "3 - Estressando a GPU com decodificação de vídeo\n";
        std::cout << "4 - Estressando a GPU com codificação de vídeo\n";
        std::cout << "5 - Estressando a GPU com operações de memória\n";
        std::cout << "6 - Estressando a GPU com todas as operações\n";
        std::cout << "Escolha: ";

        std::cin >> level;*/
        std::cout << "Entre com valores de teste válidos!";
        return 1;
    }

    // Estressar a GPU em um nível específico (por exemplo, nível 6)
    std::thread stressThread(stressGPU, level, 0); // Modificar o número da GPU conforme necessário

    // Aguardar a entrada do usuário para parar
    std::thread stopThread(stopStressInput);
    stopThread.join();

    // Finalizar o estresse
    stopStress = true;
    stressThread.join();

    return 0;
}
