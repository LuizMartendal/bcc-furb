#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <atomic>
#include <thread>
#include <chrono>
#include <nvml.h>

// Vari�veis at�micas para controle de execu��o
std::atomic<bool> running_3d{ false };
std::atomic<bool> running_mem{ false };
std::atomic<bool> running_matrix{ false };

// Encapsular todas as fun��es exportadas em extern "C"
extern "C" {

    // Fun��o para iniciar o estresse 3D
    __declspec(dllexport) void startStress3D();
    __declspec(dllexport) void stopStress3D();

    // Fun��o para iniciar o estresse de mem�ria
    __declspec(dllexport) void startStressMemory();
    __declspec(dllexport) void stopStressMemory();

    // Fun��o para iniciar a multiplica��o de matrizes
    __declspec(dllexport) void startMatrixMul(float* A, float* B, float* C, int N);
    __declspec(dllexport) void stopMatrixMul();
}

hipDeviceProp_t getCudaDeviceProp(int device) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    return prop;
}

// Kernel de estresse 3D
__global__ void stress3DKernel(float* output, int width, int height, int iterations) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = x + y * width;

    if (x < width && y < height) {
        float value = 0.0f;
        for (int i = 0; i < iterations; ++i) {
            value += sinf(x * y + i) * cosf(x - y + i);
        }
        output[idx] = value;
    }
}

// Fun��o para estressar a GPU com opera��es de 3D (realiza c�lculos simples em paralelo)
__global__ void kernel3D(float* d_out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 1 << 28) {  // Aumentando o n�mero de threads processados
        d_out[idx] = sinf((float)idx) * cosf((float)idx);  // Simples c�lculo trigonom�trico
    }
}

// Controle do estresse 3D
void startStress3D() {
    int device = 0;
    int targetGpuUsagePercentage = 50;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);

    const size_t elementSize = sizeof(float);
    size_t targetMem = static_cast<size_t>(freeMem * (targetGpuUsagePercentage / 100.0f));
    int N = targetMem / elementSize;
    float* d_out = nullptr;

    hipDeviceProp_t prop = getCudaDeviceProp(device);
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;

    nvmlInit();
    nvmlDevice_t nvmlDevice;
    nvmlDeviceGetHandleByIndex(device, &nvmlDevice);

    hipMalloc((void**)&d_out, N * elementSize);

    int blocksPerGrid;
    int threadsPerGrid;

    while (true) {
        // Monitorar uso atual da GPU
        hipMemGetInfo(&freeMem, &totalMem);

        // Monitorar uso atual da GPU
        nvmlUtilization_t utilization;
        nvmlDeviceGetUtilizationRates(nvmlDevice, &utilization);

        if (utilization.gpu < targetGpuUsagePercentage) {
            // GPU abaixo do alvo, aumentar carga
            size_t additionalMem = static_cast<size_t>(freeMem * ((targetGpuUsagePercentage - utilization.gpu) / 100.0f));
            size_t newMemUsage = std::min(additionalMem, freeMem); // Garantir que n�o exceda a mem�ria livre

            blocksPerGrid = (freeMem / elementSize + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
            blocksPerGrid = static_cast<size_t>(blocksPerGrid * ((targetGpuUsagePercentage - utilization.gpu) / 100.0f));
            threadsPerGrid = static_cast<size_t>(maxThreadsPerBlock * ((targetGpuUsagePercentage - utilization.gpu) / 100.0f));

            kernel3D << <blocksPerGrid, threadsPerGrid >> > (d_out);
            hipDeviceSynchronize();
        }
    }

    hipFree(d_out);
    nvmlShutdown();
}

void stopStress3D() {
    running_3d = false;
}

// Kernel de estresse de mem�ria
__global__ void stressMemoryKernel(float* d_mem, int size, int iterations) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        for (int i = 0; i < iterations; ++i) {
            d_mem[idx] = sinf(d_mem[idx]) + cosf(d_mem[idx]);
        }
    }
}

// Controle do estresse de mem�ria
void startStressMemory() {
    size_t totalMem = getCudaDeviceProp(0).totalGlobalMem;

    float* d_a;
    float* d_b;
    hipMalloc((void**)&d_a, totalMem);
    hipMalloc((void**)&d_b, totalMem);

    while (true) {}

    hipFree(d_a);
    hipFree(d_b);
}

void stopStressMemory() {
    running_mem = false;
}

// Kernel de multiplica��o de matrizes
__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

// Loop de multiplica��o de matrizes
void stressMatrixMulLoop(float* A, float* B, float* C, int N) {
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    while (running_matrix) {
        matrixMulKernel << <gridSize, blockSize >> > (d_A, d_B, d_C, N);
        hipDeviceSynchronize();
        hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
        std::this_thread::sleep_for(std::chrono::milliseconds(500));
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Controle da multiplica��o de matrizes
void startMatrixMul(float* A, float* B, float* C, int N) {
    running_matrix = true;
    std::thread(stressMatrixMulLoop, A, B, C, N).detach();
}

void stopMatrixMul() {
    running_matrix = false;
}

int main() {
    startStress3D();
    return 0;
}