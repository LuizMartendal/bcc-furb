#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <atomic>
#include <thread>

// Vari�vel global para controlar quando parar o estresse
std::atomic<bool> stopStress(false);

// Fun��o para obter informa��es da GPU
void getGPUInfo() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "Nenhuma GPU CUDA dispon�vel." << std::endl;
        return;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);
        std::cout << "Informa��es da GPU " << device << ":\n";
        std::cout << "  Nome: " << prop.name << std::endl;
        std::cout << "  Mem�ria Global: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Arquitetura: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  N�cleos de Processamento: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Frequ�ncia: " << prop.clockRate / 1000 << " MHz" << std::endl;
        std::cout << std::endl;
    }
}

// Fun��o para estressar a GPU com opera��es de c�pia de mem�ria
void stressMemory() {
    const int N = 1 << 28;  // Aumentando o n�mero de elementos para uma c�pia de mem�ria maior
    float* d_a, * d_b;
    float* h_a = new float[N];
    float* h_b = new float[N];

    // Aloca��o de mem�ria na GPU
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));

    // Inicializa os dados na mem�ria da CPU
    for (int i = 0; i < N; ++i) {
        h_a[i] = 1.0f;
    }

    while (!stopStress) {
        // Copia os dados para a mem�ria da GPU
        hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, d_a, N * sizeof(float), hipMemcpyDeviceToDevice);  // C�pia entre a GPU
        hipMemcpy(h_b, d_b, N * sizeof(float), hipMemcpyDeviceToHost);  // C�pia de volta para a CPU
    }

    // Libera a mem�ria alocada na GPU
    hipFree(d_a);
    hipFree(d_b);

    delete[] h_a;
    delete[] h_b;
}

// Fun��o para estressar a GPU com opera��es de mem�ria (aloca��o e desaloca��o repetidas)
void stressCopy() {
    const int N = 1 << 28;  // Aumentando o n�mero de elementos para testar mem�ria de forma mais intensiva
    float* d_data;

    while (!stopStress) {
        // Aloca e desaloca mem�ria v�rias vezes
        hipMalloc((void**)&d_data, N * sizeof(float));
        hipFree(d_data);
    }
}

// Fun��o para estressar a GPU com opera��es de 3D (realiza c�lculos simples em paralelo)
__global__ void kernel3D(float* d_out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 1 << 28) {  // Aumentando o n�mero de threads processados
        d_out[idx] = sinf((float)idx) * cosf((float)idx);  // Simples c�lculo trigonom�trico
    }
}

void stress3D() {
    const int N = 1 << 28;  // N�mero de elementos
    float* d_out;

    hipMalloc((void**)&d_out, N * sizeof(float));

    while (!stopStress) {
        // Aumentando a quantidade de blocos e threads
        kernel3D << <(N + 255) / 256, 256 >> > (d_out);
        hipDeviceSynchronize();
    }

    hipFree(d_out);
}

// Fun��o para estressar a GPU com opera��es de decodifica��o de v�deo
void stressVideoDecode() {
    const int N = 1 << 28;  // Tamanho dos dados para simular a decodifica��o
    uint8_t* d_data;

    // Aloca mem�ria na GPU
    hipMalloc((void**)&d_data, N * sizeof(uint8_t));

    while (!stopStress) {
        // Simula o processamento de v�deo decodificado (operando com os dados)
        for (int i = 0; i < N; ++i) {
            d_data[i] = (uint8_t)(i % 255);
        }
    }

    hipFree(d_data);
}

// Fun��o para estressar a GPU com opera��es de codifica��o de v�deo
void stressVideoEncode() {
    const int N = 1 << 28;  // Tamanho dos dados para simular a codifica��o
    uint8_t* d_data;

    // Aloca mem�ria na GPU
    hipMalloc((void**)&d_data, N * sizeof(uint8_t));

    while (!stopStress) {
        // Simula o processamento de v�deo codificado (operando com os dados)
        for (int i = 0; i < N; ++i) {
            d_data[i] = (uint8_t)(i % 255);
        }
    }

    hipFree(d_data);
}

// Fun��o para estressar a GPU com todas as opera��es
void stressGPU(int level, int device) {
    hipSetDevice(device); // Seleciona a GPU correta

    // Exemplo de adapta��o do estresse com base na placa
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "Usando a GPU: " << prop.name << std::endl;

    switch (level) {
    case 1:
        std::cout << "Estressando a GPU com opera��es 3D..." << std::endl;
        stress3D();
        break;
    case 2:
        std::cout << "Estressando a GPU com c�pia de mem�ria..." << std::endl;
        stressCopy();
        break;
    case 3:
        std::cout << "Estressando a GPU com decodifica��o de v�deo..." << std::endl;
        stressVideoDecode();
        break;
    case 4:
        std::cout << "Estressando a GPU com codifica��o de v�deo..." << std::endl;
        stressVideoEncode();
        break;
    case 5:
        std::cout << "Estressando a GPU com opera��es de mem�ria..." << std::endl;
        stressMemory();
        break;
    case 6:
        std::cout << "Estressando a GPU com todas as opera��es..." << std::endl;
        stress3D();
        stressCopy();
        stressVideoDecode();
        stressVideoEncode();
        stressMemory();
        break;
    default:
        std::cout << "N�mero inv�lido. Escolha um n�mero entre 1 e 6." << std::endl;
    }
}

// Fun��o para permitir ao usu�rio parar o estresse com uma tecla
void stopStressInput() {
    char stop;
    std::cout << "Digite 'q' para parar o estresse: ";
    std::cin >> stop;
    if (stop == 'q') {
        stopStress = true;
    }
}

int main(int argc, char* argv[]) {
    // Exibir as informa��es da GPU
    getGPUInfo();

    int level = std::atoi(argv[1]);

    if (level < 1 || level > 6) {
        /*std::cout << "Selecione o tipo de teste a ser feito:\n";
        std::cout << "1 - Estressando a GPU com opera��es 3D\n";
        std::cout << "2 - Estressando a GPU com c�pia de mem�ria\n";
        std::cout << "3 - Estressando a GPU com decodifica��o de v�deo\n";
        std::cout << "4 - Estressando a GPU com codifica��o de v�deo\n";
        std::cout << "5 - Estressando a GPU com opera��es de mem�ria\n";
        std::cout << "6 - Estressando a GPU com todas as opera��es\n";
        std::cout << "Escolha: ";

        std::cin >> level;*/
        std::cout << "Entre com valores de teste v�lidos!";
        return 1;
    }

    // Estressar a GPU em um n�vel espec�fico (por exemplo, n�vel 6)
    std::thread stressThread(stressGPU, level, 0); // Modificar o n�mero da GPU conforme necess�rio

    // Aguardar a entrada do usu�rio para parar
    std::thread stopThread(stopStressInput);
    stopThread.join();

    // Finalizar o estresse
    stopStress = true;
    stressThread.join();

    return 0;
}
